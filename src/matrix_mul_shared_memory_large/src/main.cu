#include <hip/hip_runtime.h>

#include <chrono>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <string>

// #include "index.cuh"

/* C = A * B
 * C = [M x N]
 * A = [M x K]
 * B = [K x N]
 */

#define SIZE_M (512 * 2)
#define SIZE_N (512 * 4)
#define SIZE_K (512 * 2)
#define BLOCK_SIZE 16

constexpr double kEpsilon = 1e-2;

void checkCudaError(hipError_t err, const char *msg);
void displayDeviceMemory();
void allocateDeviceMemory(float **d_ptr, size_t size, const char *name);
void freeDeviceMemory(void *d_ptr, const char *name);
void cleanDeviceMemory(void *d_ptr, size_t size, const char *name);
void copyToDevice(float *d_dst, const float *h_src, size_t size, const char *msg);
void copyToHost(float *h_dst, const float *d_src, size_t size, const char *msg);

__global__ void matrix_mul(const float *a, const float *b, float *c, int size_m, int size_n, int size_k) {
  unsigned int col = (blockDim.x * blockIdx.x) + threadIdx.x;
  unsigned int row = (blockDim.y * blockIdx.y) + threadIdx.y;
  unsigned int i = (row * size_n) + col;

  if (row >= size_m || col >= size_n) {
    return;
  }

  float acc = 0.0F;
  for (int k = 0; k < size_k; k++) {
    // acc += (a[(row * size_k) + k] * b[(size_n * k) + col]);
    // or
    acc = fmaf(a[(row * size_k) + k], b[(size_n * k) + col], acc);
  }
  c[i] = acc;
}

__global__ void matrix_mul_shared(const float *a, const float *b, float *c) {
  unsigned int col = (blockDim.x * blockIdx.x) + threadIdx.x;
  unsigned int row = (blockDim.y * blockIdx.y) + threadIdx.y;
  unsigned int i = (row * SIZE_N) + col;

  __shared__ float shared_a[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float shared_b[BLOCK_SIZE][BLOCK_SIZE];

  int local_row = threadIdx.y;
  int local_col = threadIdx.x;

  float acc = 0.0F;

  for (int tile = 0; tile < SIZE_K / BLOCK_SIZE; tile++) {
    int stride = tile * BLOCK_SIZE;

    // if (row >= SIZE_M || (stride + local_col) >= SIZE_K) {
    //   shared_a[local_row][local_col] = 0.0F;
    // } else {
      shared_a[local_row][local_col] = a[(row * SIZE_K) + (stride + local_col)];
    // }

    // if (col >= SIZE_N || (stride + local_row) >= SIZE_K) {
    //   shared_b[local_row][local_col] = 0.0F;
    // } else {
      shared_b[local_row][local_col] = b[((stride + local_row) * SIZE_N) + col];
    // }

    __syncthreads(); // wait until all threads finish loading

    for (int index = 0; index < BLOCK_SIZE; index++) {
      acc = fmaf(shared_a[local_row][index], shared_b[index][local_col], acc);
    }

    __syncthreads(); // wait until all threads finish computing
  }


  if (row >= SIZE_M || col >= SIZE_N) {
    return;
  }

  c[i] = acc;
}

int main(int argc, char *argv[]) {
  size_t size_a = SIZE_M * SIZE_K * sizeof(float);
  size_t size_b = SIZE_K * SIZE_N * sizeof(float);
  size_t size_c = SIZE_M * SIZE_N * sizeof(float);
  printf("Matrix multiplication: C = A * B\n");
  printf("A [%d x %d]\n", SIZE_M, SIZE_K);
  printf("B [%d x %d]\n", SIZE_K, SIZE_N);
  printf("C [%d x %d]\n", SIZE_M, SIZE_N);
  printf("\n");

  printf("Allocate Host memory\n");
  auto *h_a = (float *)malloc(size_a);
  auto *h_b = (float *)malloc(size_b);
  auto *h_c = (float *)malloc(size_c);
  auto *h_hc = (float *)malloc(size_c);
  if (h_a == nullptr || h_b == nullptr || h_c == nullptr || h_hc == nullptr) {
    fprintf(stderr, "Failed to allocate host matrix\n");
    exit(EXIT_FAILURE);
  }

  printf("Initialize Host matrix\n");
  for (int i = 0; i < (SIZE_M * SIZE_K); i++) {
    h_a[i] = ((rand() % 10) + ((rand() % 100) / 100.0));
  }
  for (int i = 0; i < (SIZE_K * SIZE_N); i++) {
    h_b[i] = ((rand() % 10) + ((rand() % 100) / 100.0));
  }
  printf("Matrix A[0][0] = %f\n", h_a[0]);
  printf("\n");

  printf("Matrix mul on Host\n");
  auto start = std::chrono::steady_clock::now();
  for (int x = 0; x < SIZE_N; x++) {
    for (int y = 0; y < SIZE_M; y++) {
      unsigned int i = (y * SIZE_N) + x;
      float acc = 0.0F;
      for (int k = 0; k < SIZE_K; k++) {
        // h_hc[i] += (h_a[(y * SIZE_K) + k] * h_b[(k * SIZE_N) + x]);
        acc = fmaf(h_a[(y * SIZE_K) + k], h_b[(k * SIZE_N) + x], acc);
      }
      h_hc[i] = acc;
    }
  }

  auto end = std::chrono::steady_clock::now();
  auto millis = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
  printf("Host matrix mul duration: %ld ms\n", millis.count());
  printf("\n");

  printf("Allocate Device memory\n");
  float *d_a;
  float *d_b;
  float *d_c;
  allocateDeviceMemory(&d_a, size_a, "d_a");
  allocateDeviceMemory(&d_b, size_b, "d_b");
  allocateDeviceMemory(&d_c, size_c, "d_c");

  printf("Copy: Host to Device\n");
  start = std::chrono::steady_clock::now();
  copyToDevice(d_a, h_a, size_a, "Failed to copy h_a to Device");
  copyToDevice(d_b, h_b, size_b, "Failed to copy h_b to Device");
  end = std::chrono::steady_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
  printf("Copy duration: %ld ms\n", duration.count());
  printf("\n");

  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 gridDim((SIZE_N + blockDim.x - 1) / blockDim.x,
               (SIZE_M + blockDim.y - 1) / blockDim.y);
  printf("Block Dim: (%d, %d, %d)\n", blockDim.x, blockDim.y, blockDim.z);
  printf("Grid Dim: (%d, %d, %d)\n", gridDim.x, gridDim.y, gridDim.z);

  printf("Launch matrix_mul kernel\n");
  start = std::chrono::steady_clock::now();
  matrix_mul<<<gridDim, blockDim>>>(d_a, d_b, d_c, SIZE_M, SIZE_N, SIZE_K);
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  millis = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
  checkCudaError(hipGetLastError(), "Failed to launch kernel");
  printf("Kernel execution duration: %ld ms\n", millis.count());
  printf("\n");

  printf("Launch matrix_mul_shared kernel\n");
  start = std::chrono::steady_clock::now();
  matrix_mul_shared<<<gridDim, blockDim>>>(d_a, d_b, d_c);
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  millis = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
  checkCudaError(hipGetLastError(), "Failed to launch kernel");
  printf("Kernel execution duration: %ld ms\n", millis.count());
  printf("\n");

  printf("Copy: Device to Host\n");
  start = std::chrono::steady_clock::now();
  copyToHost(h_c, d_c, size_c, "Failed to copy d_c to Host");
  end = std::chrono::steady_clock::now();
  duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
  printf("Copy duration: %ld µs\n", duration.count());
  printf("\n");

  printf("Verify results\n");
  for (int i = 0; i < SIZE_M * SIZE_N; i++) {
    if (fabs(h_hc[i] - h_c[i]) > kEpsilon) {
    // if (h_hc[i] != h_c[i]) {
      printf("h_hc[%d] = %f, h_c[%d] = %f\n", i, h_hc[i], i, h_c[i]);
      fprintf(stderr, "Result verification failed at %d\n", i);
      exit(EXIT_FAILURE);
    }
  }
  printf("Result verification: OK\n");
  printf("\n");

  printf("Free Device memory\n");
  freeDeviceMemory(d_a, "d_a");
  freeDeviceMemory(d_b, "d_b");
  freeDeviceMemory(d_c, "d_c");
  printf("\n");

  printf("Free Host memory\n");
  free(h_a);
  free(h_b);
  free(h_c);
  free(h_hc);

  return EXIT_SUCCESS;
}

void checkCudaError(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

void displayDeviceMemory() {
  size_t free;
  size_t total;

  hipMemGetInfo(&free, &total);

  printf("-----Device memory-----\n");
  printf("free:  %zu bytes\n", free);
  printf("total: %zu bytes\n\n", total);
}

void allocateDeviceMemory(float **d_ptr, size_t size, const char *name) {
  hipError_t err = hipMalloc((void **)d_ptr, size);
  std::string msg = std::string("Failed to allocate device memory for ") + name;
  checkCudaError(err, msg.c_str());
}

void freeDeviceMemory(void *d_ptr, const char *name) {
  hipError_t err = hipFree(d_ptr);
  std::string msg = std::string("Failed to free device memory for ") + name;
  checkCudaError(err, msg.c_str());
}

void cleanDeviceMemory(void *d_ptr, size_t size, const char *name) {
  hipError_t err = hipMemset(d_ptr, 0, size);
  std::string msg = std::string("Failed to clean memory for ") + name;
  checkCudaError(err, msg.c_str());
}

void copyToDevice(float *d_dst, const float *h_src, size_t size, const char *msg) {
  hipError_t err = hipMemcpy(d_dst, h_src, size, hipMemcpyHostToDevice);
  checkCudaError(err, msg);
}

void copyToHost(float *h_dst, const float *d_src, size_t size, const char *msg) {
  hipError_t err = hipMemcpy(h_dst, d_src, size, hipMemcpyDeviceToHost);
  checkCudaError(err, msg);
}
