#include "hip/hip_runtime.h"
#include <cstdio>

#include "kernel.h"

__global__ void helloCUDA() { printf("Hello CUDA from GPU!\n"); }

void launchHelloCUDA() {
  helloCUDA<<<1, 1>>>();
  checkCudaError(hipGetLastError(), "Failed to launch kernel");
}

void checkCudaError(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
