#include "hip/hip_runtime.h"
#include <stdio.h>

#include "kernel.h"

__global__ void helloCUDA(void) { printf("Hello CUDA from GPU!\n"); }

void launchHelloCUDA(void) {
  helloCUDA<<<1, 1>>>();
  checkCudaError(hipGetLastError(), "Failed to launch kernel");
}

void checkCudaError(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
