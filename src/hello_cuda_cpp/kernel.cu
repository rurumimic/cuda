#include "hip/hip_runtime.h"
#include <stdio.h>

#include "kernel.h"

__global__ void helloUDA(void) { printf("Hello CUDA from GPU!\n"); }

void launchHelloCUDA(void) {
  helloUDA<<<1, 1>>>();
  checkCudaError(hipGetLastError(), "Failed to launch kernel");
}

void checkCudaError(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
