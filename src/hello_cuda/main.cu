
#include <hip/hip_runtime.h>
#include <stdio.h>

void checkCudaError(hipError_t err, const char *msg);

__global__ void helloCUDA(void) { printf("Hello CUDA from GPU!\n"); }

int main(int argc, char *argv[]) {
  printf("Hello CUDA from CPU!\n");

  helloCUDA<<<1, 1>>>();
  checkCudaError(hipGetLastError(), "Failed to launch kernel");

  printf("Program completed successfully.\n");
  return 0;
}

void checkCudaError(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
