#include <hip/hip_runtime.h>

#include <chrono>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <string>

#define LENGTH 500000
#define THREADS_PER_BLOCK 256

constexpr double kEpsilon = 1e-5;

void checkCudaError(hipError_t err, const char *msg);
void displayDeviceMemory();
void allocateDeviceMemory(float **d_ptr, size_t size, const char *name);
void freeDeviceMemory(void *d_ptr, const char *name);
void cleanDeviceMemory(void *d_ptr, size_t size, const char *name);
void copyToDevice(float *d_dst, const float *h_src, size_t size, const char *msg);
void copyToHost(float *h_dst, const float *d_src, size_t size, const char *msg);

__global__ void vector_add(const float *a, const float *b, float *c, int length) {
  int i = (blockDim.x * blockIdx.x) + threadIdx.x;

  if (i < length) {
    c[i] = a[i] + b[i] + 0.0F;
  }
}

int main(int argc, char *argv[]) {
  size_t size = LENGTH * sizeof(float);
  printf("Vector length: %d\n", LENGTH);
  printf("\n");

  printf("Allocate Host memory\n");
  auto *h_a = (float *)malloc(size);
  auto *h_b = (float *)malloc(size);
  auto *h_c = (float *)malloc(size);
  auto *h_hc = (float *)malloc(size);
  if (h_a == nullptr || h_b == nullptr || h_c == nullptr || h_hc == nullptr) {
    fprintf(stderr, "Failed to allocate host vectors\n");
    exit(EXIT_FAILURE);
  }

  printf("Initialize Host vectors\n");
  for (int i = 0; i < LENGTH; i++) {
    h_a[i] = rand() / (float)RAND_MAX;
    h_b[i] = rand() / (float)RAND_MAX;
  }
  printf("\n");

  printf("Vector add on Host\n");
  auto start = std::chrono::steady_clock::now();
  for (int i = 0; i < LENGTH; i++) {
    h_hc[i] = h_a[i] + h_b[i];
  }
  auto end = std::chrono::steady_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  printf("Host vector add duration: %ld µs\n", duration.count());
  printf("\n");

  displayDeviceMemory();

  printf("Allocate Device memory\n");
  float *d_a;
  float *d_b;
  float *d_c;
  allocateDeviceMemory(&d_a, size, "d_a");
  allocateDeviceMemory(&d_b, size, "d_b");
  allocateDeviceMemory(&d_c, size, "d_c");

  printf("Cleaning Device memory\n");
  cleanDeviceMemory(d_a, size, "d_a");
  cleanDeviceMemory(d_b, size, "d_b");
  cleanDeviceMemory(d_c, size, "d_c");
  printf("\n");

  displayDeviceMemory();

  printf("Copy: Host to Device\n");
  start = std::chrono::steady_clock::now();
  copyToDevice(d_a, h_a, size, "Failed to copy h_a to Device");
  copyToDevice(d_b, h_b, size, "Failed to copy h_b to Device");
  end = std::chrono::steady_clock::now();
  duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  printf("Copy duration: %ld µs\n", duration.count());
  printf("\n");

  int blocksPerGrid = (LENGTH + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  printf("CUDA kernel: %d blocks x %d threads\n", blocksPerGrid, THREADS_PER_BLOCK);

  printf("Launch vector_add kernel\n");
  start = std::chrono::steady_clock::now();
  vector_add<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, LENGTH);
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  checkCudaError(hipGetLastError(), "Failed to launch kernel");
  printf("Kernel execution duration: %ld µs\n", duration.count());
  printf("\n");

  printf("Copy: Device to Host\n");
  start = std::chrono::steady_clock::now();
  copyToHost(h_c, d_c, size, "Failed to copy d_c to Host");
  end = std::chrono::steady_clock::now();
  duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  printf("Copy duration: %ld µs\n", duration.count());
  printf("\n");

  printf("Verify results\n");
  for (int i = 0; i < LENGTH; i++) {
    if (fabs(h_a[i] + h_b[i] - h_c[i]) > kEpsilon) {
      fprintf(stderr, "Result verification failed at %d\n", i);
      exit(EXIT_FAILURE);
    }
  }
  printf("Result verification: OK\n");
  printf("\n");

  displayDeviceMemory();

  printf("Free Device memory\n");
  freeDeviceMemory(d_a, "d_a");
  freeDeviceMemory(d_b, "d_b");
  freeDeviceMemory(d_c, "d_c");
  printf("\n");

  displayDeviceMemory();

  printf("Free Host memory\n");
  free(h_a);
  free(h_b);
  free(h_c);

  return EXIT_SUCCESS;
}

void checkCudaError(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

void displayDeviceMemory() {
  size_t free;
  size_t total;

  hipMemGetInfo(&free, &total);

  printf("-----Device memory-----\n");
  printf("free:  %zu bytes\n", free);
  printf("total: %zu bytes\n\n", total);
}

void allocateDeviceMemory(float **d_ptr, size_t size, const char *name) {
  hipError_t err = hipMalloc((void **)d_ptr, size);
  std::string msg = std::string("Failed to allocate device memory for ") + name;
  checkCudaError(err, msg.c_str());
}

void freeDeviceMemory(void *d_ptr, const char *name) {
  hipError_t err = hipFree(d_ptr);
  std::string msg = std::string("Failed to free device memory for ") + name;
  checkCudaError(err, msg.c_str());
}

void cleanDeviceMemory(void *d_ptr, size_t size, const char *name) {
  hipError_t err = hipMemset(d_ptr, 0, size);
  std::string msg = std::string("Failed to clean memory for ") + name;
  checkCudaError(err, msg.c_str());
}

void copyToDevice(float *d_dst, const float *h_src, size_t size, const char *msg) {
  hipError_t err = hipMemcpy(d_dst, h_src, size, hipMemcpyHostToDevice);
  checkCudaError(err, msg);
}

void copyToHost(float *h_dst, const float *d_src, size_t size, const char *msg) {
  hipError_t err = hipMemcpy(h_dst, d_src, size, hipMemcpyDeviceToHost);
  checkCudaError(err, msg);
}
