#include <hip/hip_runtime.h>

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <string>

#define LENGTH 50000
#define THREADS_PER_BLOCK 256

constexpr double kEpsilon = 1e-5;

void checkCudaError(hipError_t err, const char *msg);
void checkDeviceMemory();
void allocateDeviceMemory(float **d_ptr, size_t size, const char *name);
void freeDeviceMemory(void *d_ptr, const char *name);
void cleanDeviceMemory(void *d_ptr, size_t size, const char *name);
void copyToDevice(float *d_dst, const float *h_src, size_t size, const char *msg);
void copyToHost(float *h_dst, const float *d_src, size_t size, const char *msg);

__global__ void vector_add(const float *a, const float *b, float *c, int length) {
  int i = (blockDim.x * blockIdx.x) + threadIdx.x;

  if (i < length) {
    c[i] = a[i] + b[i] + 0.0F;
  }
}

int main(int argc, char *argv[]) {
  size_t size = LENGTH * sizeof(float);
  printf("Vector length: %d\n", LENGTH);

  printf("Allocate host memory\n");
  auto *h_a = (float *)malloc(size);
  auto *h_b = (float *)malloc(size);
  auto *h_c = (float *)malloc(size);

  if (h_a == nullptr || h_b == nullptr || h_c == nullptr) {
    fprintf(stderr, "Failed to allocate host vectors\n");
    exit(EXIT_FAILURE);
  }

  printf("Initialize host vectors\n");
  for (int i = 0; i < LENGTH; i++) {
    h_a[i] = rand() / (float)RAND_MAX;
    h_b[i] = rand() / (float)RAND_MAX;
  }

  checkDeviceMemory();
  printf("Allocate device memory\n");

  float *d_a;
  float *d_b;
  float *d_c;
  allocateDeviceMemory(&d_a, size, "d_a");
  allocateDeviceMemory(&d_b, size, "d_b");
  allocateDeviceMemory(&d_c, size, "d_c");

  checkDeviceMemory();

  printf("Copy: host to device\n");

  copyToDevice(d_a, h_a, size, "Failed to copy h_a to device");
  copyToDevice(d_b, h_b, size, "Failed to copy h_b to device");

  printf("\n");

  int blocksPerGrid = (LENGTH + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  printf("CUDA kernel: %d blocks x %d threads\n", blocksPerGrid, THREADS_PER_BLOCK);

  printf("Launch vector_add kernel\n");
  vector_add<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, LENGTH);
  checkCudaError(hipGetLastError(), "Failed to launch kernel");

  printf("Copy: device to host\n");
  copyToHost(h_c, d_c, size, "Failed to copy d_c to host");

  printf("Verify results\n");
  for (int i = 0; i < LENGTH; i++) {
    if (fabs(h_a[i] + h_b[i] - h_c[i]) > kEpsilon) {
      fprintf(stderr, "Result verification failed at %d\n", i);
      exit(EXIT_FAILURE);
    }
  }

  printf("Result verification: OK\n");
  printf("\n");

  printf("Cleaning device memory\n");
  cleanDeviceMemory(d_a, size, "d_a");
  cleanDeviceMemory(d_b, size, "d_b");
  cleanDeviceMemory(d_c, size, "d_c");

  checkDeviceMemory();

  printf("Free device memory\n");

  freeDeviceMemory(d_a, "d_a");
  freeDeviceMemory(d_b, "d_b");
  freeDeviceMemory(d_c, "d_c");

  checkDeviceMemory();

  free(h_a);
  free(h_b);
  free(h_c);

  printf("Program completed successfully.\n");
  return 0;
}

void checkCudaError(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

void checkDeviceMemory() {
  size_t free;
  size_t total;

  hipMemGetInfo(&free, &total);

  printf("-----Device memory-----\n");
  printf("free:  %zu bytes\n", free);
  printf("total: %zu bytes\n\n", total);
}

void allocateDeviceMemory(float **d_ptr, size_t size, const char *name) {
  hipError_t err = hipMalloc((void **)d_ptr, size);
  std::string msg = std::string("Failed to allocate device memory for ") + name;
  checkCudaError(err, msg.c_str());
}

void freeDeviceMemory(void *d_ptr, const char *name) {
  hipError_t err = hipFree(d_ptr);
  std::string msg = std::string("Failed to free device memory for ") + name;
  checkCudaError(err, msg.c_str());
}

void cleanDeviceMemory(void *d_ptr, size_t size, const char *name) {
  hipError_t err = hipMemset(d_ptr, 0, size);
  std::string msg = std::string("Failed to clean memory for ") + name;
  checkCudaError(err, msg.c_str());
}

void copyToDevice(float *d_dst, const float *h_src, size_t size, const char *msg) {
  hipError_t err = hipMemcpy(d_dst, h_src, size, hipMemcpyHostToDevice);
  checkCudaError(err, msg);
}

void copyToHost(float *h_dst, const float *d_src, size_t size, const char *msg) {
  hipError_t err = hipMemcpy(h_dst, d_src, size, hipMemcpyDeviceToHost);
  checkCudaError(err, msg);
}
